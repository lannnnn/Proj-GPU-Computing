#include <iostream>
#include <fstream>
#include <vector>
#include <sstream>
#include "cuda_impl.cuh"

#define BLK_SIZE 256

int main(int argc, char* argv[]) {

    int deviceCount = 0;
    int device;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    int block_cols = 8;
    int print = 0;

    hipEvent_t startTime, endTime;
    float elapsedTime = 0.0;

    // This will launch a grid that can maximally fill the GPU, on the default stream with kernel arguments
    int numBlocksPerSm = 0;
    // Number of threads my_kernel will be launched with
    int numThreads = BLK_SIZE;

    if (error_id != hipSuccess) {
        printf("hipGetDeviceCount returned %d\n-> %s\n",
           static_cast<int>(error_id), hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0) {
        printf("There are no available device(s) that support CUDA\n");
        return -1;
    } else {
        hipGetDevice(&device);
        printf("Detected %d CUDA Capable device(s)\n", deviceCount);
    }

    // std::string filename = "/home/shuxin.zheng/Proj-GPU-Computing/My_proj/data/weighted/TEST_matrix_weighted.el";
    // std::string filename = "/home/shuxin.zheng/Proj-GPU-Computing/My_proj/data/unweighted/0_mycielskian13.el";
    std::string filename = "/home/shuxin.zheng/Proj-GPU-Computing/My_proj/data/unweighted/seventh_graders.el";

    float tau = 0.4;

    if(argc >= 2) {
        readConfig(argc, argv, &filename, &block_cols, &tau, &print);
    }

    std::cout << "using matrix file: " << filename << std::endl;
    std::cout << "using blocksize: " << block_cols << std::endl;
    std::cout << "using tau: " << tau << std::endl;

    // COO coo = readELFileWeighted(filename);
    COO coo = readELFileUnweighted(filename);
    if(coo.rows == 0) {
        std::cout << "not acceptable matrix file" << std::endl;
        return;
    }

    CSR csr(coo.rows, coo.cols, coo.nnz);
    // csr to coo, build the rankMap at same time
    cooToCsr(coo, csr);
    // free the matrix, use csr
    coo.row_message.clear();
 
    // device memory allocation
    int* d_rowPtr;
    int* d_colIdx;
    int* d_groupList;
    int* d_resultList;
    int* d_groupSize;
    int* d_refRow;
    float* d_tau;
    GroupInfo* d_groupInfo;
    CHECK( hipMalloc((int**)&d_rowPtr, (csr.rows+1) * sizeof(int)));
    CHECK( hipMalloc((int**)&d_colIdx, csr.nnz * sizeof(int)));
    CHECK( hipMalloc((int**)&d_groupList, (csr.rows+1) * sizeof(int)));
    CHECK( hipMalloc((int**)&d_resultList, csr.rows * sizeof(int)));
    CHECK( hipMalloc((int**)&d_groupSize, sizeof(int)));
    CHECK( hipMalloc((int**)&d_refRow, ref_size * sizeof(int)));
    CHECK( hipMalloc((float**)&d_tau, sizeof(float)));
    CHECK( hipMemset(d_refRow, 0, ref_size * sizeof(int)));
    CHECK( hipMalloc((GroupInfo**)&d_groupInfo, csr.rows * sizeof(GroupInfo)));
    // data copy to GPU
    CHECK( hipMemcpy(d_rowPtr, &csr.rowPtr[0], (csr.rows+1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK( hipMemcpy(d_colIdx, &csr.colIdx[0], csr.nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK( hipMemcpy(d_tau, &tau, sizeof(float), hipMemcpyHostToDevice));

    // groupList initialized as 0..n
    int* h_groupList = (int*)malloc((csr.rows+1) * sizeof(int));
    int* h_resultList = (int*)malloc(csr.rows * sizeof(int));
    int* h_rowPtr = (int*)malloc((csr.rows+1) * sizeof(int));
    int* h_colIdx = (int*)malloc(csr.nnz * sizeof(int));
    for(int i=0; i< csr.rows; i++) {
        h_groupList[i] = i;
        h_resultList[i] = i;
    }
    h_groupList[csr.rows] = csr.rows;

    int* h_groupSize = (int*)malloc(sizeof(int));
    h_groupSize[0] = csr.rows;
    CHECK( hipMemcpy(d_groupList, h_groupList, (csr.rows+1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK( hipMemcpy(d_resultList, h_resultList, csr.rows * sizeof(int), hipMemcpyHostToDevice));
    CHECK( hipMemcpy(d_groupSize, h_groupSize, sizeof(int), hipMemcpyHostToDevice));

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, gpu_ref_grouping, numThreads, 0);
    int totalThreads = deviceProp.multiProcessorCount*numBlocksPerSm*BLK_SIZE;
    void *kernelArgs[] = {(void *)&d_rowPtr, (void *)&d_colIdx, (void *)&d_tau, (void *)&d_groupList, (void *)&d_groupInfo, 
                                (void *)&d_resultList, (void *)&d_groupSize, (void *)&d_refRow};
    dim3 dimBlock(numThreads, 1, 1);
    int grdDim = deviceProp.multiProcessorCount*numBlocksPerSm;
    if(totalThreads > csr.rows) grdDim = (csr.rows+BLK_SIZE)/BLK_SIZE;
    dim3 dimGrid(grdDim, 1, 1);
    // std::cout << "matrix size: (rows, cols, nnz) = (" << csr.rows << ", " << csr.cols << ", " << csr.nnz << ")" << std::endl;
    std::cout << "Start calculating with dimGrid " << grdDim << ", dimBlock " << numThreads << "..." << std::endl;

    hipEventCreate(&startTime);
    hipEventCreate(&endTime);
    hipEventRecord(startTime, 0);

    hipLaunchCooperativeKernel((void*)gpu_ref_grouping, dimGrid, dimBlock, kernelArgs);

    hipEventRecord(endTime, 0);
    hipEventSynchronize(startTime);
    hipEventSynchronize(endTime);
    hipEventElapsedTime(&elapsedTime, startTime, endTime);

    CHECK( hipMemcpy(h_groupList, d_groupList, (csr.rows) * sizeof(int), hipMemcpyDeviceToHost));
    // print_pointer(h_groupList, csr.rows);
    std::vector<std::vector<int>> fine_group(csr.rows+1);
    for(int i=0; i<csr.rows; i++) {
        fine_group[h_groupList[i]].push_back(i);
    }
    // print_pointer(h_resultList, csr.rows);
    if(print) {
        std::cout << "Reordered row rank:" << std::endl;
        print_vec(fine_group);
    }
    CSR new_csr(csr.rows, csr.cols, csr.nnz);
    reordering(csr, new_csr, fine_group);

    // clear the cuda memory
    CHECK( hipFree(d_rowPtr));
    CHECK( hipFree(d_colIdx));
    CHECK( hipFree(d_groupList));
    CHECK( hipFree(d_resultList));
    CHECK( hipFree(d_groupSize));
    CHECK( hipFree(d_refRow));
    CHECK( hipFree(d_tau));
    CHECK( hipFree(d_groupInfo));

    hipEventDestroy(startTime);
    hipEventDestroy(endTime);

    // new_csr.print();
    std::cout << "matrix info: nrows=" << csr.rows << ", ncols=" << csr.cols << ", nnz=" << csr.nnz << std::endl;
    std::cout << "checking for using block size: (" << block_cols << "," << block_cols << ")" << std::endl;
    std::cout << "original density: " << csr.calculateBlockDensity(block_cols, block_cols) << std::endl;
    std::cout << "new density: " << new_csr.calculateBlockDensity(block_cols, block_cols) << std::endl;
    std::cout << "Group calculation time(GPU): " << elapsedTime << " ms" << std::endl;
    return 0;
}