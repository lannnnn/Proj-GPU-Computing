#include <iostream>
#include <fstream>
#include <vector>
#include <sstream>
#include "cuda_impl.cuh"

#define BLK_SIZE 256

int main() {

    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    int block_cols = 8;

    if (error_id != hipSuccess) {
        printf("hipGetDeviceCount returned %d\n-> %s\n",
           static_cast<int>(error_id), hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0) {
        printf("There are no available device(s) that support CUDA\n");
        return -1;
    } else {
        hipSetDevice(0);
        printf("Detected %d CUDA Capable device(s)\n", deviceCount);
    }

    // std::string filename = "/home/shuxin.zheng/Proj-GPU-Computing/My_proj/data/weighted/TEST_matrix_weighted.el";
    // std::string filename = "/home/shuxin.zheng/Proj-GPU-Computing/My_proj/data/unweighted/0_mycielskian13.el";
    std::string filename = "/home/shuxin.zheng/Proj-GPU-Computing/My_proj/data/unweighted/seventh_graders.el";

    float tau = 0.6;
    // method allow inordered input data

    // COO coo = readELFileWeighted(filename);
    COO coo = readELFileUnweighted(filename);
    // COO coo = readMTXFileWeighted(filename);
    // print_matrix(coo, 1); //print matrix message

    CSR csr(coo.rows, coo.cols, coo.nnz);
    // csr to coo, build the rankMap at same time
    cooToCsr(coo, csr);
    // free the matrix, use csr
    coo.row_message.clear();
    // print_vec(label);
    // csr.print();
    // print_map(rankMap)
 
    // device memory allocation
    int* d_rowPtr;
    int* d_colIdx;
    int* d_groupList;
    int* d_resultList;
    int* d_groupSize;
    GroupInfo* d_groupInfo;
    CHECK( hipMalloc((int**)&d_rowPtr, (csr.rows+1) * sizeof(int)));
    CHECK( hipMalloc((int**)&d_colIdx, csr.nnz * sizeof(int)));
    CHECK( hipMalloc((int**)&d_groupList, (csr.rows+1) * sizeof(int)));
    CHECK( hipMalloc((int**)&d_resultList, csr.rows * sizeof(int)));
    CHECK( hipMalloc((int**)&d_groupSize, sizeof(int)));
    CHECK( hipMalloc((GroupInfo**)&d_groupInfo, csr.rows * sizeof(GroupInfo)));
    // data copy to GPU
    CHECK( hipMemcpy(d_rowPtr, &csr.rowPtr[0], (csr.rows+1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK( hipMemcpy(d_colIdx, &csr.colIdx[0], csr.nnz * sizeof(int), hipMemcpyHostToDevice));
    // csr.print();

    // groupList initialized as 0..n
    int* h_groupList = (int*)malloc((csr.rows+1) * sizeof(int));
    int* h_resultList = (int*)malloc(csr.rows * sizeof(int));
    int* h_rowPtr = (int*)malloc((csr.rows+1) * sizeof(int));
    int* h_colIdx = (int*)malloc(csr.nnz * sizeof(int));
    for(int i=0; i< csr.rows; i++) {
        h_groupList[i] = i;
        h_resultList[i] = i;
    }
    h_groupList[csr.rows] = csr.rows;

    int* h_groupSize = (int*)malloc(sizeof(int));
    h_groupSize[0] = csr.rows;
    CHECK( hipMemcpy(d_groupList, h_groupList, (csr.rows+1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK( hipMemcpy(d_resultList, h_resultList, csr.rows * sizeof(int), hipMemcpyHostToDevice));
    CHECK( hipMemcpy(d_groupSize, h_groupSize, sizeof(int), hipMemcpyHostToDevice));

    int grd_size = (csr.rows+BLK_SIZE)/BLK_SIZE;

    dim3 block_size(BLK_SIZE, 1, 1);
    dim3 grid_size(grd_size, 1, 1);
    gpu_grouping<<< grid_size, block_size>>>(d_rowPtr, d_colIdx, tau, d_groupList, d_groupInfo, d_resultList, 
                                                d_groupSize, csr.nnz, grd_size*BLK_SIZE, block_cols);
    // test<<< grid_size, block_size>>>(d_groupList, resultList);
    // copy data back
    // CHECK( hipMemcpy(h_groupSize, d_groupSize, sizeof(int), hipMemcpyDeviceToHost));
    CHECK( hipMemcpy(h_resultList, d_resultList, csr.rows * sizeof(int), hipMemcpyDeviceToHost));
    //tmp here
    CHECK( hipMemcpy(h_groupList, d_groupList, (h_groupSize[0]+1) * sizeof(int), hipMemcpyDeviceToHost));

    std::vector<std::vector<int>> fine_group(1, std::vector<int>(csr.rows));
    fine_group[0] = std::vector<int>(&h_resultList[0], &h_resultList[0] + csr.rows);
    // print_pointer(h_resultList, csr.rows);
    std::cout << "Reordered row rank: ";
    print_vec(fine_group);
    CSR new_csr(csr.rows, csr.cols, csr.nnz);
    reordering(csr, new_csr, fine_group);

    // std::cout << "new_csr.rowPtr.size() = " << new_csr.rowPtr.size() << std::endl;
    // std::cout << "new_csr.colIdx.size() = " << new_csr.colIdx.size() << std::endl;

    // new_csr.print();
    std::cout << "using matrix file: " << filename << std::endl;
    std::cout << "matrix info: nrows=" << csr.rows << ", ncols=" << csr.cols << ", nnz=" << csr.nnz << std::endl;
    std::cout << "checking for using block size: (" << block_cols << "," << block_cols << ")" << std::endl;
    std::cout << "original density: " << csr.calculateBlockDensity(block_cols, block_cols) << std::endl;
    std::cout << "new density: " << new_csr.calculateBlockDensity(block_cols, block_cols) << std::endl;
    std::cout << "GPU calculation time: " << std::endl;
    return 0;
}